#include "hip/hip_runtime.h"
﻿// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#if SIZE_MAX == UINT_MAX
typedef int ssize_t;        /* common 32 bit case */
#define SSIZE_MIN  INT_MIN
#define SSIZE_MAX  INT_MAX
#elif SIZE_MAX == ULONG_MAX
typedef long ssize_t;       /* linux 64 bits */
#define SSIZE_MIN  LONG_MIN
#define SSIZE_MAX  LONG_MAX
#elif SIZE_MAX == ULLONG_MAX
typedef long long ssize_t;  /* windows 64 bits */
#define SSIZE_MIN  LLONG_MIN
#define SSIZE_MAX  LLONG_MAX
#elif SIZE_MAX == USHRT_MAX
typedef short ssize_t;      /* is this even possible? */
#define SSIZE_MIN  SHRT_MIN
#define SSIZE_MAX  SHRT_MAX
#elif SIZE_MAX == UINTMAX_MAX
typedef intmax_t ssize_t;  /* last resort, chux suggestion */
#define SSIZE_MIN  INTMAX_MIN
#define SSIZE_MAX  INTMAX_MAX
#else
#error platform has exotic SIZE_MAX
#endif


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include "dirent.h">
#include "getopt.h"
#include <ctype.h>

size_t getline(char** lineptr, size_t* n, FILE* stream) {
	char* bufptr = NULL;
	char* p = bufptr;
	size_t size;
	int c;

	if (lineptr == NULL) {
		return -1;
	}
	if (stream == NULL) {
		return -1;
	}
	if (n == NULL) {
		return -1;
	}
	bufptr = *lineptr;
	size = *n;

	c = fgetc(stream);
	if (c == EOF) {
		return -1;
	}
	if (bufptr == NULL) {
		bufptr = (char *)malloc(128);
		if (bufptr == NULL) {
			return -1;
		}
		size = 128;
	}
	p = bufptr;
	while (c != EOF) {
		if ((p - bufptr) > (size - 1)) {
			size = size + 128;
			bufptr = (char *)realloc(bufptr, size);
			if (bufptr == NULL) {
				return -1;
			}
		}
		*p++ = c;
		if (c == '\n') {
			break;
		}
		c = fgetc(stream);
	}

	*p++ = '\0';
	*lineptr = bufptr;
	*n = size;

	return p - bufptr - 1;
}

char* trim(char* str) {
	size_t len = 0;
	char* frontp = str;
	char* endp = NULL;

	if (str == NULL) { return NULL; }
	if (str[0] == '\0') { return str; }

	len = strlen(str);
	endp = str + len;

	/* Move the front and back pointers to address the first non-whitespace
	 * characters from each end.
	 */
	while (isspace((unsigned char)*frontp)) { ++frontp; }
	if (endp != frontp)
	{
		while (isspace((unsigned char)*(--endp)) && endp != frontp) {}
	}

	if (str + len - 1 != endp)
		*(endp + 1) = '\0';
	else if (frontp != str && endp == frontp)
		*str = '\0';

	/* Shift the string so that it starts at str so that if it's dynamically
	 * allocated, we can still free it on the returned pointer.  Note the reuse
	 * of endp to mean the front of the string buffer now.
	 */
	endp = str;
	if (frontp != str)
	{
		while (*frontp) { *endp++ = *frontp++; }
		*endp = '\0';
	}


	return str;
}

__global__ void sha256_cuda(JOB** jobs, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	if (i < n) {
		SHA256_CTX ctx;
		sha256_init(&ctx);
		sha256_update(&ctx, jobs[i]->data, jobs[i]->size);
		sha256_final(&ctx, jobs[i]->digest);
	}
}

void pre_sha256() {
	// compy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}


void runJobs(JOB** jobs, int n) {
	int blockSize = 4;
	int numBlocks = (n + blockSize - 1) / blockSize;
	sha256_cuda << < numBlocks, blockSize >> > (jobs, n);
}


JOB* JOB_init(BYTE* data, long size, char* fname) {
	JOB* j;
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
	checkCudaErrors(hipMallocManaged(&(j->data), size));
	j->data = data;
	j->size = size;
	for (int i = 0; i < 64; i++)
	{
		j->digest[i] = 0xff;
	}
	strcpy(j->fname, fname);
	return j;
}


BYTE* get_file_data(char* fname, unsigned long* size) {
	FILE* f = 0;
	BYTE* buffer = 0;
	unsigned long fsize = 0;

	f = fopen(fname, "rb");
	if (!f) {
		fprintf(stderr, "get_file_data Unable to open '%s'\n", fname);
		return 0;
	}
	fflush(f);

	if (fseek(f, 0, SEEK_END)) {
		fprintf(stderr, "Unable to fseek %s\n", fname);
		return 0;
	}
	fflush(f);
	fsize = ftell(f);
	rewind(f);

	//buffer = (char *)malloc((fsize+1)*sizeof(char));
	checkCudaErrors(hipMallocManaged(&buffer, (fsize + 1) * sizeof(char)));
	fread(buffer, fsize, 1, f);
	fclose(f);
	*size = fsize;
	return buffer;
}

void print_usage() {
	printf("Usage: CudaSHA256 [OPTION] [FILE]...\n");
	printf("Calculate sha256 hash of given FILEs\n\n");
	printf("OPTIONS:\n");
	printf("\t-f FILE1 \tRead a list of files (separeted by \\n) from FILE1, output hash for each file\n");
	printf("\t-h       \tPrint this help\n");
	printf("\nIf no OPTIONS are supplied, then program reads the content of FILEs and outputs hash for each FILEs \n");
	printf("\nOutput format:\n");
	printf("Hash following by two spaces following by file name (same as sha256sum).\n");
	printf("\nNotes:\n");
	printf("Calculations are performed on GPU, each seperate file is hashed in its own thread\n");
}

int main(int argc, char** argv) {
	int i = 0, n = 0;
	size_t len;
	unsigned long temp;
	char* a_file = 0, * line = 0;
	BYTE* buff;
	char option, index;
	ssize_t read;
	JOB** jobs;

	// parse input
	while ((option = getopt(argc, argv, "hf:")) != -1)
		switch (option) {
		case 'h':
			print_usage();
			break;
		case 'f':
			a_file = optarg;
			break;
		default:
			break;
		}


	if (a_file) {
		FILE* f = 0;
		f = fopen(a_file, "r");
		if (!f) {
			fprintf(stderr, "Unable to open %s\n", a_file);
			return 0;
		}

		for (n = 0; getline(&line, &len, f) != -1; n++) {}
		checkCudaErrors(hipMallocManaged(&jobs, n * sizeof(JOB*)));
		fseek(f, 0, SEEK_SET);

		n = 0;
		read = getline(&line, &len, f);
		while (read != -1) {
			//printf("%s\n", line);
			read = getline(&line, &len, f);
			line = trim(line);
			buff = get_file_data(line, &temp);
			jobs[n++] = JOB_init(buff, temp, line);
		}

		pre_sha256();
		runJobs(jobs, n);

	}
	else {
		// get number of arguments = files = jobs
		n = argc - optind;
		if (n > 0) {

			checkCudaErrors(hipMallocManaged(&jobs, n * sizeof(JOB*)));

			// iterate over file list - non optional arguments
			for (i = 0, index = optind; index < argc; index++, i++) {
				buff = get_file_data(argv[index], &temp);
				jobs[i] = JOB_init(buff, temp, argv[index]);
			}

			pre_sha256();
			runJobs(jobs, n);
		}
	}

	hipDeviceSynchronize();
	print_jobs(jobs, n);
	hipDeviceReset();
	return 0;
}
